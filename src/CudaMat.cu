#include "CudaMat.cuh"

CudaMat::CudaMat(const cv::Mat& mat) {
    hipMalloc((void**)&this->device_data, mat.rows * mat.cols * CV_ELEM_SIZE(mat.type()));
    hipMemcpy(this->device_data, mat.data, mat.rows * mat.cols * CV_ELEM_SIZE(mat.type()), hipMemcpyHostToDevice);
    this->rows = mat.rows;
    this->cols = mat.cols;
    this->type = mat.type();
}

CudaMat::CudaMat(int rows, int cols, int type) {
    hipMalloc((void**)&this->device_data, rows * cols * CV_ELEM_SIZE(type));
    this->rows = rows;
    this->cols = cols;
    this->type = type;
}

CudaMat::~CudaMat() {
    hipFree(this->device_data);
}

cv::Mat CudaMat::to_host() {
    cv::Mat mat(this->rows, this->cols, this->type);
    hipMemcpy(mat.data, this->device_data, this->rows * this->cols * CV_ELEM_SIZE(this->type), hipMemcpyDeviceToHost);
    return mat;
}
